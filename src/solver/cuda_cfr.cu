#include "hip/hip_runtime.h"
#include "solver/cuda_cfr.h"
#include "solver/cuda_func.h"
#include "ranges/RiverRangeManager.h"

void cuda_error(hipError_t error, const char *file, int line) {
    if(error != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(error), file, line);
        exit(EXIT_FAILURE);
    }
}

#define CHECK_ERROR(error) (cuda_error(error, __FILE__, __LINE__))

template<class T>
void copy_to_device(T *dev, T *host, int n, bool print=false) {
    if(!dev || !host || n <= 0) return;
    size_t size = n * sizeof(T);
    CHECK_ERROR(hipMemcpy(dev, host, size, hipMemcpyHostToDevice));
    if(!print) return;
    print_data(host, n);
    print_data_kernel<<<1, 1>>>(dev, n);
    hipDeviceSynchronize();
}

int max_malloc_len(int left, int right, int group_size = 1) {
    int mid = 0, size = group_size * sizeof(float);
    float *p = nullptr;
    while(left < right) {
        mid = (left + right + 1) >> 1;// 靠右
        if(hipMalloc(&p, mid * size) == hipSuccess) {
            hipFree(p);
            left = mid;
        }
        else right = mid - 1;
    }
    return left;
}

void CudaCFR::leaf_cfv(int player) {
    Timer timer;
    int opp = 1 - player, offset = player == P0 ? 0 : hand_size[P0];
    int my_hand = hand_size[player], opp_hand = hand_size[opp];
    int size = node_cnt[FOLD_TYPE];
    int block = block_size(size);
    clear_prob_sum(size);
    fold_cfv_kernel<<<block, LANE_SIZE>>>(
        player, size, dev_leaf_node, dev_prob_sum, my_hand, opp_hand,
        dev_hand_card_ptr[opp], dev_hand_hash_ptr[opp], dev_same_hand_idx+offset
    );
    hipDeviceSynchronize();
    // printf("fold_cfv:%zd ms\n", timer.ms(true));
    
    size = node_cnt[SHOWDOWN_TYPE];
    block = block_size(size);
    clear_prob_sum(size);
    sd_cfv_kernel<<<block, LANE_SIZE>>>(
        player, size, dev_leaf_node+sd_offset, dev_prob_sum, my_hand, opp_hand,
        dev_hand_card_ptr[player], dev_hand_card_ptr[opp], n_card
    );
    hipDeviceSynchronize();
    // printf("sd_cfv:%zd ms\n", timer.ms());
}

CudaCFR::~CudaCFR() {
    if(dev_root_cfv) CHECK_ERROR(hipFree(dev_root_cfv));
    if(dev_hand_card) CHECK_ERROR(hipFree(dev_hand_card));
    if(dev_hand_hash) CHECK_ERROR(hipFree(dev_hand_hash));
    if(dev_nodes) CHECK_ERROR(hipFree(dev_nodes));
    if(dev_leaf_node) CHECK_ERROR(hipFree(dev_leaf_node));
    for(float *p : dev_data) {
        if(p) CHECK_ERROR(hipFree(p));
    }
    if(dev_prob_sum) CHECK_ERROR(hipFree(dev_prob_sum));
    for(int *p : dev_strength) {
        if(p) CHECK_ERROR(hipFree(p));
    }
}

void CudaCFR::set_cfv_and_offset(DFSNode &node, int player, float *&cfv, int &offset) {
    if(player == -1) player = node.player;// 向上连接同玩家节点
    int p_idx = node.parent_p0_idx, act_idx = node.parent_p0_act;// 向上连接P0
    if(player != P0) {// 向上连接P1
        p_idx = node.parent_p1_idx;
        act_idx = node.parent_p1_act;
    }
    if(p_idx == -1) {
        cfv = root_cfv_ptr[player];
        offset = root_prob_ptr[player] - root_cfv_ptr[player];
    }
    else {
        if(player != dfs_node[p_idx].player) throw runtime_error("player mismatch");
        cfv = dev_data[dfs_idx_map[p_idx]] + cfv_offset(hand_size[player], act_idx);
        offset = reach_prob_to_cfv(dfs_node[p_idx].n_act, hand_size[player]);
    }
}

size_t CudaCFR::init_player_node() {
    size_t total = 0, size = 0, node_size = n_player_node * sizeof(Node);
    vector<Node> cpu_node(n_player_node);// 与cuda内存对应
    CHECK_ERROR(hipMalloc(&dev_nodes, node_size));
    total += node_size;
    dev_data = vector<float*>(n_player_node, nullptr);
    dfs_idx_map = vector<int>(dfs_idx, -1);
    slice_offset = vector<vector<int>>(N_PLAYER);
    int mem_idx = 0;
    for(int i = 0; i < N_PLAYER; i++) {// 枚举player
        for(vector<int> &nodes : slice[i]) {// 枚举slice
            slice_offset[i].push_back(mem_idx);
            for(int idx : nodes) {// 枚举node
                DFSNode &node = dfs_node[idx];
                Node &target = cpu_node[mem_idx];// cpu存储位置
                target.n_act = node.n_act;
                set_cfv_and_offset(node, -1, target.parent_cfv, target.parent_offset);
                size = get_size(node.n_act, hand_size[node.player]) * sizeof(float);
                CHECK_ERROR(hipMalloc(&target.data, size));
                if(target.data == nullptr) throw runtime_error("malloc error");
                total += size;
                dev_data[mem_idx] = target.data;
                dfs_idx_map[idx] = mem_idx++;
            }
        }
        slice_offset[i].push_back(mem_idx);
    }
    CHECK_ERROR(hipMemcpy(dev_nodes, cpu_node.data(), node_size, hipMemcpyHostToDevice));
    return total;
}

size_t CudaCFR::init_leaf_node() {
    size_t node_size = n_leaf_node * sizeof(CudaLeafNode);
    vector<CudaLeafNode> cpu_node(n_leaf_node);// 与cuda内存对应
    CHECK_ERROR(hipMalloc(&dev_leaf_node, node_size));
    int mem_idx = 0;
    for(int t = 0; t < N_LEAF_TYPE; t++) {
        for(int i = 0; i < leaf_node_dfs[t].size(); i++) {
            DFSNode &node = dfs_node[leaf_node_dfs[t][i]];
            CudaLeafNode &target = cpu_node[mem_idx++];// cpu存储位置
            target.val = ev[t][i];
            target.offset_prob_sum = i * n_card;
            set_cfv_and_offset(node, P0, target.data_p0, target.offset_p0);
            set_cfv_and_offset(node, P1, target.data_p1, target.offset_p1);
            int j = decode_idx0(node.info), k = decode_idx1(node.info);
            size_t info = init_board;
            if(t == FOLD_TYPE) {
                if(j != -1) info |= 1LL << poss_card[j];
                if(k != -1) info |= 1LL << poss_card[k];
                target.info = (int *)info;
            }
            else {
                if(j == -1) info = 0;
                else if(k == -1) info = j;
                else info = tril_idx(j, k);
                target.info = dev_strength[info];
            }
        }
    }
    CHECK_ERROR(hipMemcpy(dev_leaf_node, cpu_node.data(), node_size, hipMemcpyHostToDevice));
    sd_offset = leaf_node_dfs[FOLD_TYPE].size();
    ev.clear();
    return node_size;
}

size_t CudaCFR::init_memory() {
    size_t total = 0;
    int n = root_prob.size();
    root_cfv = vector<float>(n, 0);
    size_t size = ((n << 1)) * sizeof(float);// cfv + prob
    CHECK_ERROR(hipMalloc(&dev_root_cfv, size));
    total += size;
    root_cfv_ptr[P0] = dev_root_cfv;
    root_cfv_ptr[P1] = dev_root_cfv + hand_size[P0];
    root_prob_ptr[P0] = root_cfv_ptr[P0] + n;
    root_prob_ptr[P1] = root_prob_ptr[P0] + hand_size[P0];
    clear_root_cfv();
    copy_to_device(root_prob_ptr[P0], root_prob.data(), n);

    vector<int> temp_hand_card = hand_card;
    vector<size_t> temp_hand_hash = hand_hash;
    // [P0,P1,P0]
    temp_hand_card.insert(temp_hand_card.end(), hand_card.begin(), hand_card.begin()+(hand_size[P0]<<1));
    temp_hand_hash.insert(temp_hand_hash.end(), hand_hash.begin(), hand_hash.begin()+hand_size[P0]);
    n = temp_hand_card.size();
    size = (n + same_hand_idx.size()) * sizeof(int);// [P0,P1,P0] + [P0,P1]
    CHECK_ERROR(hipMalloc(&dev_hand_card, size));
    total += size;
    copy_to_device(dev_hand_card, temp_hand_card.data(), n);
    dev_same_hand_idx = dev_hand_card + n;
    copy_to_device(dev_same_hand_idx, same_hand_idx.data(), same_hand_idx.size());
    
    n = temp_hand_hash.size();
    size = n * sizeof(size_t);
    CHECK_ERROR(hipMalloc(&dev_hand_hash, size));
    total += size;
    copy_to_device(dev_hand_hash, temp_hand_hash.data(), n);
    dev_hand_card_ptr[P0] = dev_hand_card;
    dev_hand_card_ptr[P1] = dev_hand_card + (hand_size[P0]<<1);
    dev_hand_hash_ptr[P0] = dev_hand_hash;
    dev_hand_hash_ptr[P1] = dev_hand_hash + hand_size[P0];
    
    total += init_player_node();
    total += init_strength_table();
    total += init_leaf_node();
    
    // FOLD_TYPE,SHOWDOWN_TYPE,共用dev_prob_sum
    int len = max(node_cnt[FOLD_TYPE], node_cnt[SHOWDOWN_TYPE]);
    size = len * n_card * sizeof(float);
    CHECK_ERROR(hipMalloc(&dev_prob_sum, size));
    total += size;
    return total;
}

size_t CudaCFR::init_strength_table() {
    SliceCFR::init_strength_table();
    int n = strength.size();
    size_t total = 0, size = 0;
    dev_strength = vector<int*>(n, nullptr);
    for(int i = 0; i < n; i++) {
        const RiverCombs *p0_comb = strength[i][P0].data, *p1_comb = strength[i][P1].data;
        int p0_size = strength[i][P0].size, p1_size = strength[i][P1].size, d = 0;
        vector<int> data(2+((p0_size+p1_size)<<1));
        data[d++] = 2 + (p0_size<<1);
        data[d++] = data.size();
        for(int j = 0; j < p0_size; j++) {
            data[d++] = p0_comb[j].rank;
            data[d++] = p0_comb[j].reach_prob_index;
        }
        for(int j = 0; j < p1_size; j++) {
            data[d++] = p1_comb[j].rank;
            data[d++] = p1_comb[j].reach_prob_index;
        }
        size = data.size() * sizeof(int);
        CHECK_ERROR(hipMalloc(&dev_strength[i], size));
        total += size;
        copy_to_device(dev_strength[i], data.data(), data.size());
    }
    strength.clear();
    rrm.clear();
    return total;
}

size_t CudaCFR::estimate_tree_size() {
    for(int i = 0; i < N_TYPE; i++) node_cnt[i] = 0;
    if(tree == nullptr) return 0;
    size_t size = _estimate_tree_size(tree->getRoot());
    n_leaf_node = node_cnt[FOLD_TYPE] + node_cnt[SHOWDOWN_TYPE];
    n_player_node = node_cnt[N_LEAF_TYPE+P0] + node_cnt[N_LEAF_TYPE+P1];
    size *= sizeof(float);
    size += n_leaf_node * sizeof(CudaLeafNode);
    size += n_player_node * sizeof(Node);
    size += max(node_cnt[FOLD_TYPE], node_cnt[SHOWDOWN_TYPE]) * n_card * sizeof(float);
    return size;
}

void CudaCFR::_reach_prob(int player, bool avg_strategy) {
    vector<int>& offset = slice_offset[player];
    int n = offset.size() - 1, size = 0, block = 0, n_hand = hand_size[player];
    for(int i = 0; i < n; i++) {
        size = offset[i+1] - offset[i];
        block = block_size(size);
        if(avg_strategy) reach_prob_avg_kernel<<<block, LANE_SIZE>>>(dev_nodes+offset[i], size, n_hand);
        else reach_prob_kernel<<<block, LANE_SIZE>>>(dev_nodes+offset[i], size, n_hand);
        hipDeviceSynchronize();
    }
}

void CudaCFR::_rm(int player, bool avg_strategy) {
    int size = node_cnt[N_LEAF_TYPE + player];
    int block = block_size(size);
    Node *node = dev_nodes + slice_offset[player][0];
    if(avg_strategy) rm_avg_kernel<<<block, LANE_SIZE>>>(node, size, hand_size[player]);
    else rm_kernel<<<block, LANE_SIZE>>>(node, size, hand_size[player]);
    hipDeviceSynchronize();
}

void CudaCFR::clear_data(int player) {
    int size = node_cnt[N_LEAF_TYPE + player];
    int block = block_size(size);
    clear_data_kernel<<<block, LANE_SIZE>>>(dev_nodes+slice_offset[player][0], size, hand_size[player]);
    hipDeviceSynchronize();
}

void CudaCFR::clear_prob_sum(int len) {
    CHECK_ERROR(hipMemset(dev_prob_sum, 0, len * n_card * sizeof(float)));
    hipDeviceSynchronize();
}

void CudaCFR::clear_root_cfv() {
    CHECK_ERROR(hipMemset(dev_root_cfv, 0, root_cfv.size() * sizeof(float)));
    hipDeviceSynchronize();
}

void CudaCFR::step(int iter, int player, int task) {
    Timer timer;
    int opp = 1 - player, my_hand = hand_size[player], size = 0, block = 0;
    _reach_prob(opp, task != CFR_TASK);
    size_t t1 = timer.ms(true);

    leaf_cfv(player);
    size_t t2 = timer.ms(true);

    if(task == CFR_TASK) {
        size = n_player_node;
        block = block_size(size);
        discount_data_kernel<<<block, LANE_SIZE>>>(dev_nodes, size, my_hand, pos_coef, neg_coef, coef);
        hipDeviceSynchronize();
    }
    size_t t3 = timer.ms(true);
    vector<int>& offset = slice_offset[player];
    for(int i = offset.size()-2; i >= 0; i--) {
        size = offset[i+1] - offset[i];
        block = block_size(size);
        if(task == EXP_TASK) best_cfv_kernel<<<block, LANE_SIZE>>>(dev_nodes+offset[i], size, my_hand);
        else cfv_kernel<<<block, LANE_SIZE>>>(dev_nodes+offset[i], size, my_hand);
        hipDeviceSynchronize();
    }
    size_t t4 = timer.ms();
    printf("%zd\t%zd\t%zd\t%zd\n", t1, t2, t3, t4);
}

void CudaCFR::post_process() {
    int n = root_cfv.size();
    CHECK_ERROR(hipMemcpy(root_cfv.data(), dev_root_cfv, n * sizeof(float), hipMemcpyDeviceToHost));
    // print_data(root_cfv.data(), n);
    // print_data_kernel<<<1, 1>>>(dev_root_cfv, n);
    // hipDeviceSynchronize();
}

vector<vector<float>> CudaCFR::get_avg_strategy(int idx) {
    DFSNode &node = dfs_node[idx];
    int n_hand = hand_size[node.player], n_act = node.n_act;
    int size = n_act * n_hand, i = 0, h = 0, j = 0;
    float *dev = dev_data[dfs_idx_map[idx]] + (size << 1), sum = 0, uni = 1.0 / n_act;
    vector<float> strategy_sum(size);// [n_act,n_hand]
    CHECK_ERROR(hipMemcpy(strategy_sum.data(), dev, size * sizeof(float), hipMemcpyDeviceToHost));
    vector<vector<float>> strategy(n_hand, vector<float>(n_act));// [n_hand,n_act]
    for(h = 0; h < n_hand; h++) {
        sum = 0;
        for(i = h; i < size; i += n_hand) sum += strategy_sum[i];
        if(sum == 0) {
            for(j = 0; j < n_act; j++) strategy[h][j] = uni;
        }
        else {
            for(j = 0, i = h; j < n_act; j++, i += n_hand) strategy[h][j] = strategy_sum[i] / sum;
        }
    }
    return strategy;
}
vector<vector<float>> CudaCFR::get_ev(int idx) {
    return {};
}
void CudaCFR::cfv_to_ev() {}
