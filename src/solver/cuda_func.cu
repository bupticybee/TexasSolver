#include "hip/hip_runtime.h"
#include "solver/cuda_cfr.h"
#include "solver/cuda_func.h"
#include ""

__host__ __device__ void print_data(int *arr, int n) {
    if(arr != nullptr && n > 0) {
        printf("%d", arr[0]);
        for(int i = 1; i < n; i++) printf(",%d", arr[i]);
    }
    printf("\n");
}
__host__ __device__ void print_data(size_t *arr, int n) {
    if(arr != nullptr && n > 0) {
        printf("%llx", arr[0]);
        for(int i = 1; i < n; i++) printf(",%llx", arr[i]);
    }
    printf("\n");
}
__host__ __device__ void print_data(float *arr, int n) {
    if(arr != nullptr && n > 0) {
        printf("%.2f", arr[0]);
        for(int i = 1; i < n; i++) printf(",%.2f", arr[i]);
    }
    printf("\n");
}
__global__ void print_data_kernel(int *arr, int n) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i == 0) print_data(arr, n);
}
__global__ void print_data_kernel(size_t *arr, int n) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i == 0) print_data(arr, n);
}
__global__ void print_data_kernel(float *arr, int n) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i == 0) print_data(arr, n);
}

__global__ void clear_data_kernel(Node *node, int size, int n_hand) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= size) return;
    node += i;
    size = get_size(node->n_act, n_hand);
    float *data = node->data;
    for(i = 0; i < size; i++) data[i] = 0;
}

// 不同节点之间独立
__global__ void rm_avg_kernel(Node *node, int size, int n_hand) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= size) return;
    node += i;
    size = node->n_act * n_hand;
    int h = 0, sum_offset = size << 1;
    float *data = node->data + (size << 1);// strategy_sum
    float sum = 0;
    for(h = 0; h < n_hand; h++) {
        sum = 0;
        for(i = h; i < size; i += n_hand) sum += data[i];
        data[sum_offset+h] = sum;
    }
}
__global__ void rm_kernel(Node *node, int size, int n_hand) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= size) return;
    node += i;
    size = node->n_act * n_hand;
    int h = 0, sum_offset = size * 3;
    float *data = node->data + size;// regret_sum
    float sum = 0;
    for(h = 0; h < n_hand; h++) {
        sum = 0;
        for(i = h; i < size; i += n_hand) sum += max(0.0f, data[i]);
        data[sum_offset+h] = sum;
    }
}

// 上层slice传递到本层slice
__global__ void reach_prob_avg_kernel(Node *node, int size, int n_hand) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= size) return;
    node += i;
    int n_act = node->n_act;
    size = n_act * n_hand;
    int h = 0, sum_offset = size << 1;
    float *data = node->data + (size << 1);// strategy_sum
    float *parent_prob = node->parent_cfv + node->parent_offset, temp = 0;
    for(h = 0; h < n_hand; h++) {
        if(data[sum_offset+h] == 0) {// 1/n_act
            temp = parent_prob[h] / n_act;
            for(i = h; i < size; i += n_hand) data[size+i] = temp;
        }
        else {
            temp = parent_prob[h] / data[sum_offset+h];
            for(i = h; i < size; i += n_hand) data[size+i] = temp * data[i];
        }
    }
}
__global__ void reach_prob_kernel(Node *node, int size, int n_hand) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= size) return;
    node += i;
    int n_act = node->n_act;
    size = n_act * n_hand;
    int h = 0, rp_offset = size << 1, sum_offset = rp_offset + size;
    float *data = node->data + size;// regret_sum
    float *parent_prob = node->parent_cfv + node->parent_offset, temp = 0;
    for(h = 0; h < n_hand; h++) {
        if(data[sum_offset+h] == 0) {// 1/n_act
            temp = parent_prob[h] / n_act;
            for(i = h; i < size; i += n_hand) data[rp_offset+i] = temp;
        }
        else {
            temp = parent_prob[h] / data[sum_offset+h];
            for(i = h; i < size; i += n_hand) data[rp_offset+i] = temp * max(0.0f, data[i]);
        }
    }
}

// 叶子节点向上层slice聚合,调用前需要清零上层slice的cfv
// same_hand_idx:player same_hand_idx
// hand_hash,hand_card:init opp   [P0,P1,P0]
__global__ void fold_cfv_kernel(int player, int size, CudaLeafNode *node, float *opp_prob_sum, int my_hand, int opp_hand, int *hand_card, size_t *hand_hash, int *same_hand_idx) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= size) return;
    node += i;
    opp_prob_sum += node->offset_prob_sum;
    size_t board = (size_t)node->info;
    float *cfv = nullptr, *opp_reach = nullptr, val = node->val;
    float prob_sum = 0, temp = 0;
    if(player == P0) {
        cfv = node->data_p0, opp_reach = node->data_p1 + node->offset_p1;
    }
    else {
        cfv = node->data_p1, opp_reach = node->data_p0 + node->offset_p0;
        val = -val;
    }
    for(i = 0; i < opp_hand; i++) {
        if(hand_hash[i] & board) continue;// 对方手牌与公共牌冲突
        temp = opp_reach[i];
        opp_prob_sum[hand_card[i]] += temp;// card1
        opp_prob_sum[hand_card[i+opp_hand]] += temp;// card2
        prob_sum += temp;
    }
    hand_hash += opp_hand;// ptr of player
    hand_card += (opp_hand << 1);
    for(i = 0; i < my_hand; i++) {
        if(hand_hash[i] & board) {
            // cfv[i] = 0;// 与公共牌冲突，cfv为0
            continue;
        }
        temp = same_hand_idx[i] != -1 ? opp_reach[same_hand_idx[i]] : 0;// 重复计算的部分
        temp = (prob_sum - opp_prob_sum[hand_card[i]] - opp_prob_sum[hand_card[i+my_hand]] + temp) * val;
        atomicAdd(cfv+i, temp);
    }
}

// showdown
__global__ void sd_cfv_kernel(int player, int size, CudaLeafNode *node, float *opp_prob_sum, int my_hand, int opp_hand, int *my_card, int *opp_card, int n_card) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= size) return;// 总任务数
    node += i;
    opp_prob_sum += node->offset_prob_sum;
    float *cfv = nullptr, *opp_reach = nullptr;
    float prob_sum = 0, temp = 0;
    int j = 0, size_j = 0, h = 0, s = 0, *strength_data = node->info;
    // strength_data:2+size0,2+size0+size1,sorted_data
    // i,size for player
    // j,size_j for opp
    if(player == P0) {
        i = 2, size_j = strength_data[1];
        size = j = strength_data[0];
        cfv = node->data_p0, opp_reach = node->data_p1 + node->offset_p1;
    }
    else {
        j = 2, size = strength_data[1];
        size_j = i = strength_data[0];
        cfv = node->data_p1, opp_reach = node->data_p0 + node->offset_p0;
    }
    // strength_data += 2;
    for(; i < size; i += 2) {// strength值变小,己方手牌变强
        s = strength_data[i];
        for(; j < size_j && strength_data[j] > s; j += 2) {// (胜过对方条件下)找到对方的最强手牌
            h = strength_data[j+1];
            temp = opp_reach[h];
            opp_prob_sum[opp_card[h]] += temp;// card1
            opp_prob_sum[opp_card[h+opp_hand]] += temp;// card2
            prob_sum += temp;
        }
        h = strength_data[i+1];
        temp = (prob_sum - opp_prob_sum[my_card[h]] - opp_prob_sum[my_card[h+my_hand]]) * node->val;
        atomicAdd(cfv+h, temp);
    }
    prob_sum = 0;
    for(h = 0; h < n_card; h++) opp_prob_sum[h] = 0;
    i -= 2, j -= 2;
    if(player == P0) {
        size_j = size;
        size = 2;
    }
    else {
        size = size_j;
        size_j = 2;
    }
    for(; i >= size; i -= 2) {// strength值变大,己方手牌变弱
        s = strength_data[i];
        for(; j >= size_j && strength_data[j] < s; j -= 2) {// (败给对方条件下)找到对方的最弱手牌
            h = strength_data[j+1];
            temp = opp_reach[h];
            opp_prob_sum[opp_card[h]] += temp;// card1
            opp_prob_sum[opp_card[h+opp_hand]] += temp;// card2
            prob_sum += temp;
        }
        h = strength_data[i+1];
        temp = (opp_prob_sum[my_card[h]] + opp_prob_sum[my_card[h+my_hand]] - prob_sum) * node->val;
        atomicAdd(cfv+h, temp);
    }
}

// 本层slice向上层slice聚合,上层cfv需要先清零
// 子节点cfv中选最大值
__global__ void best_cfv_kernel(Node *node, int size, int n_hand) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= size) return;
    node += i;
    size = node->n_act * n_hand;
    int h = 0;
    float *parent_cfv = node->parent_cfv, *cfv = node->data, val = 0;
    for(h = 0; h < n_hand; h++) {
        val = cfv[h];// 第一个
        for(i = h+n_hand; i < size; i += n_hand) val = max(val, cfv[i]);
        atomicAdd(parent_cfv+h, val);
    }
    for(i = 0; i < size; i++) cfv[i] = 0;// 清零cfv
}
// 子节点cfv加权求和
__global__ void cfv_kernel(Node *node, int size, int n_hand) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= size) return;
    node += i;
    int n_act = node->n_act;
    size = n_act * n_hand;
    int h = 0, sum_offset = size << 2;
    float *parent_cfv = node->parent_cfv, *cfv = node->data, val = 0;
    float *regret_sum = cfv + size;
    for(h = 0; h < n_hand; h++) {
        val = 0;
        if(cfv[sum_offset+h] == 0) {
            for(i = h; i < size; i += n_hand) val += cfv[i];
            val /= n_act;// uniform strategy
        }
        else {
            for(i = h; i < size; i += n_hand) {
                val += cfv[i] * max(0.0f, regret_sum[i]);
            }
            val /= cfv[sum_offset+h];
        }
        atomicAdd(parent_cfv+h, val);
        for(i = h; i < size; i += n_hand) regret_sum[i] += cfv[i] - val;// 更新regret_sum
        val = 0;
        for(i = h; i < size; i += n_hand) val += max(0.0f, regret_sum[i]);
        cfv[sum_offset+h] = val;// 求和
    }
    for(i = 0; i < size; i++) cfv[i] = 0;// 清零cfv
}

__global__ void discount_data_kernel(Node *node, int size, int n_hand, float pos_coef, float neg_coef, float coef) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= size) return;
    node += i;
    size = node->n_act * n_hand;
    float *regret_sum = node->data + size, *strategy_sum = regret_sum + size;
    for(i = 0; i < size; i++) {
        regret_sum[i] *= regret_sum[i] > 0 ? pos_coef : neg_coef;
        strategy_sum[i] = strategy_sum[i] * coef + strategy_sum[size+i];
    }
}
